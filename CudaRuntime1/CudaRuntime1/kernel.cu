#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void add(int a[], int b[], int c[]) {
    for (int i = 0; i < sizeof(a); i++) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    const int N = 5;
    int a[N] = { 2, 2, 2, 1, 11 };
    int b[N] = { 1, 5, 3, 2, 7 };
    int c[N] = { 0 };
    int size = N * sizeof(int);
    int* d_a = 0;
    int* d_b = 0;
    int* d_c = 0;

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

    add << <1, N >> > (d_a, d_b, d_c);

    hipDeviceSynchronize();

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("{ 2, 2, 2, 1, 11 } + { 1, 5, 3, 2, 7 } = { %d, %d, %d, %d, %d }\n",
        c[0], c[1], c[2], c[3], c[4]);

    hipDeviceReset();

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}